#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <iostream>
#include "blur.h"
#include "Color.h"

__global__ void separateChannels(const uchar4* const inputImageRGBA , int numRows , int numCols , unsigned char* const redChannel , 
                      unsigned char* const greenChannel , unsigned char* const blueChannel);

__global__
void recombineChannels(const unsigned char* const redChannel , const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols);

__global__
void gaussianBlur(const unsigned char* const inputChannel , unsigned char* const outputChannel, int numRows , int numCols , const float* const filter, int filterWidth);

__global__ 
void boxBlur(unsigned char* input_image, unsigned char* output_image, int width, int height, int fsize);

Blurrer * Blurrer::instance = NULL;

Blurrer::Blurrer()
{
}

Blurrer * Blurrer::factory()
{
	if(Blurrer::instance == NULL)
	{
		Blurrer::instance = new Blurrer();
	}
	return Blurrer::instance;
	
}

void Blurrer::BlurPixels(xlColor *pixels,int rows, int cols, int filterWidth)
{
	numRows = rows;
	numCols = cols;
	allocateMemory(pixels);
	makeFilter(filterWidth);
	wrapperBlurrer();
	hipMemcpy(pixels, d_outputImageRGBA , sizeof(uchar4) * numRows * numCols, hipMemcpyDeviceToHost);
	resetBlurrer();
}

void Blurrer::allocateMemory(xlColor *pixels)
{
	const size_t numPixels = numRows * numCols;
	h_inputImageRGBA  = (uchar4 *)pixels;
	//h_outputImageRGBA = (uchar4 *)pixels;
	hipMalloc(&d_inputImageRGBA, sizeof(uchar4) * numPixels);
	hipMalloc(&d_outputImageRGBA, sizeof(uchar4) * numPixels);
	hipMemset(d_outputImageRGBA, 0, numPixels * sizeof(uchar4));
	hipMemcpy(d_inputImageRGBA, h_inputImageRGBA, sizeof(uchar4) * numPixels, hipMemcpyHostToDevice);
	hipMalloc(&d_redBlurred,    sizeof(unsigned char) * numPixels);
	hipMalloc(&d_greenBlurred,  sizeof(unsigned char) * numPixels);
	hipMalloc(&d_blueBlurred,   sizeof(unsigned char) * numPixels);
	hipMemset(d_redBlurred,   0, sizeof(unsigned char) * numPixels);
	hipMemset(d_greenBlurred, 0, sizeof(unsigned char) * numPixels);
	hipMemset(d_blueBlurred,  0, sizeof(unsigned char) * numPixels);
	hipMalloc(&d_red,   sizeof(unsigned char) * numRows * numCols);
	hipMalloc(&d_green, sizeof(unsigned char) * numRows * numCols);
	hipMalloc(&d_blue,  sizeof(unsigned char) * numRows * numCols);
	

}

void Blurrer::makeFilter(int fWidth)
{
	const int blurKernelWidth = 9;
	const float blurKernelSigma = 2.;
	filterWidth = blurKernelWidth;

	//create and fill the filter we will convolve with
	h_filter = new float[blurKernelWidth * blurKernelWidth];
	float filterSum = 0.f; //for normalization
	for (int r = -blurKernelWidth/2; r <= blurKernelWidth/2; ++r) 
	{
		for (int c = -blurKernelWidth/2; c <= blurKernelWidth/2; ++c) 
		{
			float filterValue = expf( -(float)(c * c + r * r) / (2.f * blurKernelSigma * blurKernelSigma));
      			h_filter[(r + blurKernelWidth/2) * blurKernelWidth + c + blurKernelWidth/2] = filterValue;
      			filterSum += filterValue;
    		}	
  	}

  	float normalizationFactor = 1.f / filterSum;

  	for (int r = -blurKernelWidth/2; r <= blurKernelWidth/2; ++r) 
	{
    	for (int c = -blurKernelWidth/2; c <= blurKernelWidth/2; ++c) 
		{
			h_filter[(r + blurKernelWidth/2) * blurKernelWidth + c + blurKernelWidth/2] *= normalizationFactor;
    	}
  	}	
	
	hipMalloc(&d_filter , filterWidth * filterWidth * sizeof(float));
	hipMemcpy(d_filter , h_filter , filterWidth * filterWidth * sizeof(float) , hipMemcpyHostToDevice);
}

/* Wrapper to call the blur kernel.
 * Initializes the block and grid dimensions and calls a series of 3 kernels  : 
 * seperate Channels , gaussianBlur : once for each colour channel.
 * recombineChannels
 */
void Blurrer::wrapperBlurrer()
{
	const int BLOCK_WIDTH =  32;
	const dim3 blockSize(BLOCK_WIDTH , BLOCK_WIDTH);
	const dim3 gridSize((numCols/BLOCK_WIDTH) + 1 , (numRows/BLOCK_WIDTH) + 1 );
	
	separateChannels<<<gridSize,blockSize>>>(d_inputImageRGBA , numRows , numCols, d_red, d_green , d_blue);
        hipDeviceSynchronize(); hipGetLastError();
  
	gaussianBlur<<<gridSize , blockSize>>>(d_red , d_redBlurred, numRows, numCols, d_filter, filterWidth);
	gaussianBlur<<<gridSize , blockSize>>>(d_green , d_greenBlurred, numRows, numCols, d_filter, filterWidth);
	gaussianBlur<<<gridSize , blockSize>>>(d_blue , d_blueBlurred, numRows, numCols, d_filter, filterWidth);
	hipDeviceSynchronize(); hipGetLastError();
  
	recombineChannels<<<gridSize, blockSize>>>(d_redBlurred , d_greenBlurred , d_blueBlurred , d_outputImageRGBA , numRows, numCols);
	hipDeviceSynchronize(); hipGetLastError();
}

void Blurrer::filterBlur (unsigned char* input_image, unsigned char* output_image, int width, int height, int filterSize) {

    unsigned char* dev_input;
    unsigned char* dev_output;
    hipMalloc( (void**) &dev_input, width*height*3*sizeof(unsigned char));
    hipMemcpy( dev_input, input_image, width*height*3*sizeof(unsigned char), hipMemcpyHostToDevice );
 
    hipMalloc( (void**) &dev_output, width*height*3*sizeof(unsigned char));

    dim3 blockDims(512,1,1);
    dim3 gridDims((unsigned int) ceil((double)(width*height*3/blockDims.x)), 1, 1 );


    boxBlur<<<gridDims, blockDims>>>(dev_input, dev_output, width, height, filterSize);
           

    hipMemcpy(output_image, dev_output, width*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost );

    hipFree(dev_input);
    hipFree(dev_output);

}

void Blurrer::resetBlurrer()
{
	hipFree(d_red);
	hipFree(d_green);
	hipFree(d_blue);
	hipFree(d_filter);
	hipFree(d_redBlurred);	
	hipFree(d_greenBlurred);
	hipFree(d_blueBlurred);
	hipFree(d_inputImageRGBA);
	hipFree(d_outputImageRGBA);
	delete [] h_filter;
	h_inputImageRGBA = NULL;
	//h_outputImageRGBA = NULL;
	d_inputImageRGBA = NULL;
	d_outputImageRGBA = NULL;
	d_red = NULL;
	d_green = NULL;
	d_blue = NULL;
	h_filter = NULL;
	d_filter = NULL;
	d_redBlurred = NULL;
	d_greenBlurred = NULL; 	
	d_blueBlurred = NULL;
	filterWidth = 0;
	numRows = 0;
	numCols = 0;
}

Blurrer::~Blurrer()
{
	
}

__global__
void gaussianBlur(const unsigned char* const inputChannel , unsigned char* const outputChannel, int numRows , int numCols, const float* const filter, int filterWidth)
{
	int half_width = filterWidth/2;
	float image_value = 0.0f;
	float blur_value = 0.0f;
	float computed_value = 0.0f;
	int row = 0;
	int column = 0;
	const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);
	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows) {
		return;
	}

	for(row = -half_width ; row <= half_width ; ++row) {
		for(column = -half_width; column <= half_width ; ++column) {
			int image_r = min(max(thread_2D_pos.y + row, 0), (numRows - 1));
			int image_c = min(max(thread_2D_pos.x + column, 0), (numCols - 1));
			image_value = static_cast<float>(inputChannel[(image_r) * numCols + (image_c)]);
			blur_value = filter[(row + half_width) * filterWidth + (column + half_width)];
			computed_value += image_value * blur_value;
		}       
	}
    
	outputChannel[thread_1D_pos] = static_cast<char>(computed_value);	
}

/* This kernel takes in an image where each pixel is represented as a uchar4 and splits
 * it into three color channels . 
 */
__global__ void separateChannels(const uchar4* const inputImageRGBA , int numRows , int numCols , unsigned char* const redChannel , 
                      unsigned char* const greenChannel , unsigned char* const blueChannel)
{
	const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)	
	{
		return;
	}

	redChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].x;
	greenChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].y;
	blueChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].z;

}

/* Recombines the three colour channels to form a single output coloured image */
__global__
void recombineChannels(const unsigned char* const redChannel , const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
	const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);
	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
	{
		return;
	}

	unsigned char red   = redChannel[thread_1D_pos];
	unsigned char green = greenChannel[thread_1D_pos];
	unsigned char blue  = blueChannel[thread_1D_pos];
	uchar4 outputPixel = make_uchar4(red, green, blue, 255);
	outputImageRGBA[thread_1D_pos] = outputPixel;
}

__global__
void boxBlur(unsigned char* input_image, unsigned char* output_image, int width, int height, int fsize) {

    const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
    int x = offset % width;
    int y = (offset-x)/width;
    if(offset < width*height) {

        float output_red = 0;
        float output_green = 0;
        float output_blue = 0;
        int hits = 0;
        for(int ox = -fsize; ox < fsize+1; ++ox) {
            for(int oy = -fsize; oy < fsize+1; ++oy) {
                if((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) {
                    const int currentoffset = (offset+ox+oy*width)*3;
                    output_red += input_image[currentoffset]; 
                    output_green += input_image[currentoffset+1];
                    output_blue += input_image[currentoffset+2];
                    hits++;
                }
            }
        }
        output_image[offset*3] = output_red/hits;
        output_image[offset*3+1] = output_green/hits;
        output_image[offset*3+2] = output_blue/hits;
    }
}