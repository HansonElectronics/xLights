#include "hip/hip_runtime.h"

#include "CudaEffectDataTypes.h"
#include "CudaRenderBufferComputeData.cuh"

#include "../../PixelBuffer.h"
#include "../../RenderBuffer.h"

#include <hip/hip_runtime.h>
#include <>

__global__ void apply_box_blur(int height, int width, unsigned char* buffer, unsigned char* out) {
    int i, j;

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < 2 || col < 2 || row >= height -3 || col >= width -3 ) return ;

    float v = 1.0 / 9.0;
    float kernel[3][3] = { {v,v,v},
                        {v,v,v},
                        {v,v,v} };

    float sum0 = 0.0;
    float sum1 = 0.0;
    float sum2 = 0.0;
    for (i = -1; i <= 1; i++)
    {
        for (j = -1; j <= 1; j++)
        {
            // matrix multiplication with kernel with every color plane
            sum0 = sum0 + (float)kernel[i + 1][j + 1] * buffer[((row + i) * width + (col + j)) * 3 + 0];
            sum1 = sum1 + (float)kernel[i + 1][j + 1] * buffer[((row + i) * width + (col + j)) * 3 + 1];
            sum2 = sum2 + (float)kernel[i + 1][j + 1] * buffer[((row + i) * width + (col + j)) * 3 + 2];
        }
    }
    out[(row * width + col) * 3 + 0] = (unsigned char)sum0;
    out[(row * width + col) * 3 + 1] = (unsigned char)sum1;
    out[(row * width + col) * 3 + 2] = (unsigned char)sum2;
};

CudaPixelBufferComputeData::CudaPixelBufferComputeData() {
}

CudaPixelBufferComputeData::~CudaPixelBufferComputeData() {
}

CudaRenderBufferComputeData::CudaRenderBufferComputeData(RenderBuffer *rb, CudaPixelBufferComputeData *pbd) :
    renderBuffer(rb), 
    pixelBufferData(pbd) {

}

CudaRenderBufferComputeData::~CudaRenderBufferComputeData() {
    pixelBufferData = nullptr;
}

void CudaRenderBufferComputeData::commit() {

}

void CudaRenderBufferComputeData::waitForCompletion() {

}

void CudaRenderBufferComputeData::bufferResized() {

}

bool CudaRenderBufferComputeData::blur(int radius) {
    if ((renderBuffer->BufferHt < (radius * 2)) || (renderBuffer->BufferWi < (radius * 2)) || ((renderBuffer->BufferWi * renderBuffer->BufferHt) < 1024)) {
        // Smallish buffer, overhead of sending to GPU will be more than the gain
        return false;
    }
    unsigned char* d_buffer;
    unsigned char* d_out;
    const size_t numPixels = renderBuffer->BufferWi * renderBuffer->BufferHt;

    // allocate space for device copies
    hipMalloc((void**)&d_buffer, numPixels * 4 * sizeof(unsigned char));
    hipMalloc((void**)&d_out, numPixels * 4 * sizeof(unsigned char));
    //
    // Copy inputs to device
    hipMemcpy(d_buffer, renderBuffer->GetPixels(), numPixels * 4 * sizeof(unsigned char), hipMemcpyHostToDevice);
    //
    //// perform the Box blur and store the resulting pixels in the output buffer
    dim3 blockSize(16, 16, 1);
    dim3 gridSize((numPixels*4)/blockSize.x, (numPixels*4)/blockSize.y, 1);
    apply_box_blur <<<gridSize, blockSize>>> (radius, radius, d_buffer, d_out);
    //copy output from device
    hipMemcpy(renderBuffer->GetPixels(), d_out, numPixels * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(d_buffer);
    hipFree(d_out);
    return true;
}

bool CudaRenderBufferComputeData::rotoZoom(GPURenderUtils::RotoZoomSettings &settings) {
    if ((renderBuffer->BufferWi * renderBuffer->BufferHt) < 256) {
        // Smallish buffer, overhead of sending to GPU will be more than the gain
        return false;
    }

    RotoZoomData data;
    data.width = renderBuffer->BufferWi;
    data.height = renderBuffer->BufferHt;
    
    data.offset = settings.offset;
    data.xrotation = settings.xrotation;
    data.xpivot = settings.xpivot;
    data.yrotation = settings.yrotation;
    data.ypivot = settings.ypivot;
    data.zrotation = settings.zrotation;
    data.zoom = settings.zoom;
    data.zoomquality = settings.zoomquality;
    data.pivotpointx = settings.pivotpointx;
    data.pivotpointy = settings.pivotpointy;

    for (auto &c : settings.rotationorder) {
        switch (c) {
            case 'X':
                if (data.xrotation != 0 && data.xrotation != 360) {
                   // callRotoZoomFunction(CudaComputeUtilities::INSTANCE.xrotateFunction, data);
                }
                break;
            case 'Y':
                if (data.yrotation != 0 && data.yrotation != 360) {
                    //callRotoZoomFunction(CudaComputeUtilities::INSTANCE.yrotateFunction, data);
                }
                break;
            case 'Z':
                if (data.zrotation != 0.0 || data.zoom != 1.0) {
                    //callRotoZoomFunction(CudaComputeUtilities::INSTANCE.zrotateFunction, data);
                }
                break;
        }
    }
    return true;
}

CudaRenderBufferComputeData *CudaRenderBufferComputeData::getCudaRenderBufferComputeData(RenderBuffer *b) {
    return static_cast<CudaRenderBufferComputeData*>(b->gpuRenderData);
}